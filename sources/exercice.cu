#include "hip/hip_runtime.h"
#include "exercice.h"

/**
 * L’ensemble de Julia est d´efini par la suite
zn+1 = zn^2 + c (1)
o`u zn et c sont des nombre complexes. Le nombre c est consid´er´e comme constant et vaut ici c = −0.5 + 0.6i. La
suite est initialis´ee `a z0 qui correspond aux coordonn´ees d’un pixel d’une image o`u la partie r´eelle correspond `a
l’axe x et la partie imaginaire `a l’axe y. L’objectif est de g´en´erer une image de N colonnes et M lignes o`u chaque
pixel d’indice (i, j) indique si la suite de l’Equation 1 converge ou pas. On consid`ere que le suite converge d´es lors
que |zn|
2 < 2, ∀n ≤ 100, o`u |zn| est le module de zn. On se concentre sur la zone x ∈ (−1.5, +1.5) en horizontal et
y ∈ (−1.0, +1.0) en vertical. On note C le nombre de canaux dans l’image g´en´er´ee, valant 1 dans les deux premi`eres
´etapes, et 3 (pour les composantes rouge, vert et bleu) dans la troisi`eme et derni`ere ´etape.
Le fichier exercice.h contient la fonction get ptr pour acc´eder au contenu de l’image. Dans tous les exercices,
la m´emoire doit ˆetre allou´ee sur GPU en utilisant la fonction hipMallocPitch. Les noyaux CUDA 2D sont tous
lanc´es avec BLOCK SIZE×BLOCK SIZE threads par block.
 */

/*
Le premi`ere ´etape consiste `a g´en´erer une image en noir et blanc o`u un pixel est blanc (= 1.0) si la suite converge,
ou noir (= 0.0) sinon.
1. Dans le fichier exercice.cu, compl´eter le noyau CUDA 2D kernel generate1 qui traite un pixel de l’image
binaire g´en´er´ee avec C = 1.
*/
__global__ void kernel_generate1(int N, int M, int C, int pitch, float *img)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < M)
    {
        float x = -1.5 + 3.0 * i / N;
        float y = -1.0 + 2.0 * j / M;
        int n = 0;
        while (n < 100 && x * x + y * y < 4.0)
        {
            float x_new = x * x - y * y - 0.5;
            float y_new = 2.0 * x * y + 0.6;
            x = x_new;
            y = y_new;
            n++;
        }
        img[j * pitch + i] = n < 100 ? 1.0 : 0.0;
    }
}

/**
 * Dans le fichier exercice.cu, compl ́eter la fonction generate1 pour allouer de la m ́emoire sur GPU, lancer
le noyau CUDA avec BLOCK SIZE×BLOCK SIZE threads par block, rapatrier les donn ́ees sur CPU, lib ́erer la
m ́emoire allou ́ee sur GPU, puis retourner le r ́esultat
 */
float *generate1(int N, int M, int C)
{
    float *img = nullptr;
    float *d_img = nullptr;
    size_t pitch;
    hipMallocPitch(&d_img, &pitch, N * sizeof(float), M);
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((N + block_size.x - 1) / block_size.x, (M + block_size.y - 1) / block_size.y);
    kernel_generate1<<<grid_size, block_size>>>(N, M, C, pitch / sizeof(float), d_img);
    img = new float[N * M];
    hipMemcpy2D(img, N * sizeof(float), d_img, pitch, N * sizeof(float), M, hipMemcpyDeviceToHost);
    hipFree(d_img);
    return img;
}

__global__ void kernel_generate2(int N, int M, int C, int pitch, float *img)
{
}

float *generate2(int N, int M, int C)
{
    return nullptr;
}

__global__ void kernel_generate3(int N, int M, int C, int pitch, float *img)
{
}

float *generate3(int N, int M, int C)
{
    return nullptr;
}
